#include "hip/hip_runtime.h"
/*
NV12ToARGB color space conversion CUDA kernel

This sample uses CUDA to perform a simple NV12 (YUV 4:2:0 planar) 
source and converts to output in ARGB format
*/

#include "CudaPostProcessing.h"

__constant__ uint32 constAlpha;

#define MUL(x,y)    (x*y)

__constant__ float  constHueColorSpaceMat[9];


extern "C"
hipError_t  UpdateConstantMemory(float *hueCSC)
{
	hipDeviceptr_t d_constHueCSC, d_constAlpha;
	unsigned int d_cscBytes, d_alphaBytes;

	// First grab the global device pointers from the CUBIN
	//hipModuleGetGlobal(&d_constHueCSC,  &d_cscBytes  , module, "constHueColorSpaceMat");
	//hipModuleGetGlobal(&d_constAlpha ,  &d_alphaBytes, module, "constAlpha"           );

	hipError_t error = hipSuccess;

	// Copy the constants to video memory
	hipMemcpyHtoD( d_constHueCSC, reinterpret_cast<const void *>(hueCSC), d_cscBytes);
	
	cutilDrvCheckMsg("hipMemcpyHtoD (d_constHueCSC) copy to Constant Memory failed");

	uint32 cudaAlpha = ((uint32)0xff<< 24);

	hipMemcpyHtoD( constAlpha, reinterpret_cast<const void *>(&cudaAlpha), d_alphaBytes);
	
	cutilDrvCheckMsg("hipMemcpyHtoD (constAlpha) copy to Constant Memory failed");

	return error;
}

extern "C"
void SetColorSpaceMatrix(eColorSpace CSC, float *hueCSC)
{
	float hueSin = 0.0f;
	float hueCos = 1.0f;

	//optimize !!!
	if (CSC == ITU601) {
		//CCIR 601
		hueCSC[0] = 1.1644f;
		hueCSC[1] = hueSin * 1.5960f;
		hueCSC[2] = hueCos * 1.5960f;
		hueCSC[3] = 1.1644f;
		hueCSC[4] = (hueCos * -0.3918f) - (hueSin * 0.8130f);
		hueCSC[5] = (hueSin *  0.3918f) - (hueCos * 0.8130f);  
		hueCSC[6] = 1.1644f;
		hueCSC[7] = hueCos *  2.0172f;
		hueCSC[8] = hueSin * -2.0172f;
	} else if (CSC == ITU709) {
		//CCIR 709
		hueCSC[0] = 1.0f;
		hueCSC[1] = hueSin * 1.57480f;
		hueCSC[2] = hueCos * 1.57480f;
		hueCSC[3] = 1.0;
		hueCSC[4] = (hueCos * -0.18732f) - (hueSin * 0.46812f);
		hueCSC[5] = (hueSin *  0.18732f) - (hueCos * 0.46812f);  
		hueCSC[6] = 1.0f;
		hueCSC[7] = hueCos *  1.85560f;
		hueCSC[8] = hueSin * -1.85560f;
	}
}



__device__ void YUV2RGB(uint32 *yuvi, float *red, float *green, float *blue)
{
	float luma, chromaCb, chromaCr;

	// Prepare for hue adjustment
	luma     = (float)yuvi[0];
	chromaCb = (float)((int32)yuvi[1] - 512.0f);
	chromaCr = (float)((int32)yuvi[2] - 512.0f);

	// Convert YUV To RGB with hue adjustment
	*red  = MUL(luma,     constHueColorSpaceMat[0]) + 
		MUL(chromaCb, constHueColorSpaceMat[1]) + 
		MUL(chromaCr, constHueColorSpaceMat[2]);
	*green= MUL(luma,     constHueColorSpaceMat[3]) + 
		MUL(chromaCb, constHueColorSpaceMat[4]) + 
		MUL(chromaCr, constHueColorSpaceMat[5]);
	*blue = MUL(luma,     constHueColorSpaceMat[6]) + 
		MUL(chromaCb, constHueColorSpaceMat[7]) + 
		MUL(chromaCr, constHueColorSpaceMat[8]);
}


__device__ uint32 RGBAPACK_8bit(float red, float green, float blue, uint32 alpha)
{
	uint32 ARGBpixel = 0;

	// Clamp final 10 bit results
	red   = min(max(red,   0.0f), 255.0f);
	green = min(max(green, 0.0f), 255.0f);
	blue  = min(max(blue,  0.0f), 255.0f);

	// Convert to 8 bit unsigned integers per color component
	ARGBpixel = (((uint32)blue ) | 
		(((uint32)green) << 8)  | 
		(((uint32)red  ) << 16) | (uint32)alpha);

	return  ARGBpixel;
}

__device__ uint32 RGBAPACK_10bit(float red, float green, float blue, uint32 alpha)
{
	uint32 ARGBpixel = 0;

	// Clamp final 10 bit results
	red   = min(max(red,   0.0f), 1023.f);
	green = min(max(green, 0.0f), 1023.f);
	blue  = min(max(blue,  0.0f), 1023.f);

	// Convert to 8 bit unsigned integers per color component
	ARGBpixel = (((uint32)blue  >> 2) | 
		(((uint32)green >> 2) << 8)  | 
		(((uint32)red   >> 2) << 16) | (uint32)alpha);

	return  ARGBpixel;
}


// CUDA kernel for outputing the final ARGB output from NV12;
extern "C"
__global__ void Passthru_drvapi(uint32 *srcImage,	uint32 nSourcePitch, 
								uint32 *dstImage,	uint32 nDestPitch,
								uint32 width,		uint32 height)
{
	int32 x, y;
	uint32 yuv101010Pel[2];
	uint32 processingPitch = ((width) + 63) & ~63;
	uint32 dstImagePitch   = nDestPitch >> 2;
	uint8 *srcImageU8     = (uint8 *)srcImage;

	processingPitch = nSourcePitch;

	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	y = blockIdx.y *  blockDim.y       +  threadIdx.y;

	if (x >= width)
		return; //x = width - 1;
	if (y >= height)
		return; // y = height - 1;

	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values
	yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]);
	yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]);

	// this steps performs the color conversion
	float luma[2];

	luma[0]   =  (yuv101010Pel[0]        & 0x00FF );	
	luma[1]   =  (yuv101010Pel[1]        & 0x00FF );	

	// Clamp the results to RGBA
	dstImage[y * dstImagePitch + x     ] = RGBAPACK_8bit(luma[0], luma[0], luma[0], constAlpha);
	dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_8bit(luma[1], luma[1], luma[1], constAlpha);
}


// CUDA kernel for outputing the final ARGB output from NV12;
extern "C"
__global__ void CudaNV12ToARGBKernel(	uint32 *srcImage,		uint32 nSourcePitch, 
										uint32 *dstImage,		uint32 nDestPitch,
										uint32 width,			uint32 height)
{
	int32 x, y;
	uint32 yuv101010Pel[2];
	uint32 processingPitch = ((width) + 63) & ~63;
	uint32 dstImagePitch   = nDestPitch >> 2;
	uint8 *srcImageU8     = (uint8 *)srcImage;

	processingPitch = nSourcePitch;

	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	y = blockIdx.y *  blockDim.y       +  threadIdx.y;

	if (x >= width)
		return; //x = width - 1;
	if (y >= height)
		return; // y = height - 1;

	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values
	yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
	yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

	uint32 chromaOffset    = processingPitch * height;
	int32 y_chroma = y >> 1;

	if (y & 1)  // odd scanline ?
	{
		uint32 chromaCb;
		uint32 chromaCr;

		chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
		chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

		if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
		{
			chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
			chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
		}

		yuv101010Pel[0] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		yuv101010Pel[1] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}
	else
	{
		yuv101010Pel[0] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[0] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		yuv101010Pel[1] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[1] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}

	// this steps performs the color conversion
	uint32 yuvi[6];
	float red[2], green[2], blue[2];

	yuvi[0] =  (yuv101010Pel[0] &   COLOR_COMPONENT_MASK    );	
	yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK); 
	yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

	yuvi[3] =  (yuv101010Pel[1] &   COLOR_COMPONENT_MASK    );	
	yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK); 
	yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

	// YUV to RGB Transformation conversion
	YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
	YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

	// Clamp the results to RGBA
	dstImage[y * dstImagePitch + x     ] = RGBAPACK_10bit(red[0], green[0], blue[0], constAlpha);
	dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_10bit(red[1], green[1], blue[1], constAlpha);
}

void		CudaNV12ToARGB(	uint32 *srcImage,		uint32 nSourcePitch, 
						   uint32 *dstImage,		uint32 nDestPitch,
						   uint32 width,			uint32 height)
{
	//
}